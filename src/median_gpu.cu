#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/random.h>
#include <thrust/generate.h>
#include <thrust/detail/type_traits.h>

const int BLOCKDIM = 32;
const int MAX_WINDOW = 11;
__device__ const int FILTER_SIZE = 9;
__device__ const int FILTER_HALFSIZE = FILTER_SIZE >> 1;

__device__ void sort_quick(float *x, int left, int right) {
      int i = left, j = right;
      float tmp;
      float pivot = x[(left + right) / 2];
      while (i <= j) {
            while (x[i] < pivot)
                  i++;
            while (x[j] > pivot)
                  j--;
            if (i <= j) {
                  tmp = x[i];
                  x[i] = x[j];
                  x[j] = tmp;
                  i++;
                  j--;
            }
      };
      if (left < j)
            sort_quick(x, left, j);
      if (i < right)
            sort_quick(x, i, right);
}




__device__ void sort_bubble(float *x, int n_size) {
	for (int i = 0; i < n_size - 1; i++) {
		for(int j = 0; j < n_size - i - 1; j++) {
			if (x[j] > x[j+1]) {
				float temp = x[j];
				x[j] = x[j+1];
				x[j+1] = temp;
			}
		}
	}
}

__device__ void sort_linear(float *x, int n_size) {
	for (int i = 0; i < n_size-1; i++) {
		int min_idx = i;
		for (int j = i + 1; j < n_size; j++) {
			if(x[j] < x[min_idx])
				min_idx = j;
		}
		float temp = x[min_idx];
		x[min_idx] = x[i];
		x[i] = temp;
	}
}




__device__ int index(int x, int y, int width) 
{
	return (y * width) + x;
}

__device__ int clamp(int value, int bound) 
{
	if (value < 0) {
		return 1;
	}
	if (value < bound) {
		return value;
	}
	return bound - 1;
}

__global__ void median_filter_2d(unsigned char* input, unsigned char* output, int width, int height)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if((x<width) && (y<height))
	{
		const int color_tid = y * width + x;
		float xs[MAX_WINDOW*MAX_WINDOW];
		int xs_size = 0;

		for (int x_iter = x - FILTER_HALFSIZE; x_iter <= x + FILTER_HALFSIZE; x_iter ++)
		 {
			for (int y_iter = y - FILTER_HALFSIZE; y_iter <= y + FILTER_HALFSIZE; y_iter++)
			 {
				if (0<=x_iter && x_iter < width && 0 <= y_iter && y_iter < height)
				{
					xs[xs_size++] = input[y_iter * width + x_iter];
				}
			}
		}
		sort_bubble(xs,xs_size);
		//sort_linear(xs,xs_size);
		//sort_quick(xs,0,xs_size);
		output[color_tid] = xs[xs_size/2];
	}
}

/*

__global__ void median_filter_2d_sm(unsigned char* input, unsigned char* output, int width, int height)
{
	__shared__ int sharedPixels[BLOCKDIM + FILTER_SIZE][BLOCKDIM + FILTER_SIZE];
	
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	int xBlockLimit_max = blockDim.x - FILTER_HALFSIZE - 1;
	int yBlockLimit_max = blockDim.y - FILTER_HALFSIZE - 1;
	int xBlockLimit_min = FILTER_HALFSIZE;
	int yBlockLimit_min = FILTER_HALFSIZE;

	if (threadIdx.x > xBlockLimit_max && threadIdx.y > yBlockLimit_max) {
		int i = index(clamp(x + FILTER_HALFSIZE,width), clamp(y + FILTER_HALFSIZE,height), width);
	    	unsigned int pixel = input[i];
		sharedPixels[threadIdx.x + 2*FILTER_HALFSIZE][threadIdx.y + 2*FILTER_HALFSIZE] = pixel;
	}
	if (threadIdx.x > xBlockLimit_max && threadIdx.y < yBlockLimit_min) {
		int i = index(clamp(x + FILTER_HALFSIZE,width), clamp(y - FILTER_HALFSIZE,height), width);
		unsigned int pixel = input[i];
		sharedPixels[threadIdx.x + 2*FILTER_HALFSIZE][threadIdx.y] = pixel;
	}
	if (threadIdx.x < xBlockLimit_min && threadIdx.y > yBlockLimit_max) {
		int i = index(clamp(x - FILTER_HALFSIZE,width), clamp(y + FILTER_HALFSIZE,height), width);
		unsigned int pixel = input[i];
		sharedPixels[threadIdx.x][threadIdx.y + 2*FILTER_HALFSIZE] = pixel;
	}
	if (threadIdx.x < xBlockLimit_min && threadIdx.y < yBlockLimit_min) {
		int i = index(clamp(x - FILTER_HALFSIZE,width), clamp(y - FILTER_HALFSIZE,height), width);
		unsigned int pixel = input[i];
		sharedPixels[threadIdx.x][threadIdx.y] = pixel;
	}
	if (threadIdx.x < xBlockLimit_min) {
		int i = index(clamp(x - FILTER_HALFSIZE,width), clamp(y,height), width);
		unsigned int pixel = input[i];
		sharedPixels[threadIdx.x][threadIdx.y + FILTER_HALFSIZE] = pixel;
	}
	if (threadIdx.x > xBlockLimit_max) {
		int i = index(clamp(x + FILTER_HALFSIZE,width), clamp(y,height), width);
		unsigned int pixel = input[i];
		sharedPixels[threadIdx.x + 2*FILTER_HALFSIZE][threadIdx.y + FILTER_HALFSIZE] = pixel;
	}
	if (threadIdx.y < yBlockLimit_min) {
		int i = index(clamp(x,width), clamp(y - FILTER_HALFSIZE,height), width);
		unsigned int pixel = input[i];
		sharedPixels[threadIdx.x + FILTER_HALFSIZE][threadIdx.y] = pixel;
	}
	if (threadIdx.y > yBlockLimit_max) {
		int i = index(clamp(x,width), clamp(y + FILTER_HALFSIZE,height), width);
		unsigned int pixel = input[i];
		sharedPixels[threadIdx.x + FILTER_HALFSIZE][threadIdx.y + 2*FILTER_HALFSIZE] = pixel;
	}
	int i = index(x, y, width);
	unsigned int pixel = input[i];
	sharedPixels[threadIdx.x + FILTER_HALFSIZE][threadIdx.y + FILTER_HALFSIZE] = pixel;

	__syncthreads();

	if((x<width) && (y<height))
	{
		const int color_tid = y * width + x;
		float xs[MAX_WINDOW*MAX_WINDOW];
		int xs_size = 0;

		for (int x_iter = 0; x_iter < FILTER_SIZE; x_iter ++) 
		{
			for (int y_iter = 0; y_iter < FILTER_SIZE; y_iter++) 
			{
				if (0<=x_iter && x_iter < width && 0 <= y_iter && y_iter < height) 
				{
					xs[xs_size++] = sharedPixels[threadIdx.x + x_iter][threadIdx.y + y_iter];
				}
			}
		}
		sort_vec(xs,xs_size);
		output[color_tid] = static_cast<unsigned char>(xs[xs_size/2]);
	}
}
*/

void median_filter_wrapper(const cv::Mat& input, cv::Mat& output)
{
	unsigned char *d_input, *d_output;

	
	hipError_t cudaStatus;	
	
	cudaStatus = hipMalloc<unsigned char>(&d_input,input.rows*input.cols);
	checkCudaErrors(cudaStatus);	
	cudaStatus = hipMalloc<unsigned char>(&d_output,output.rows*output.cols);
	checkCudaErrors(cudaStatus);

	cudaStatus = hipMemcpy(d_input,input.ptr(),input.rows*input.cols,hipMemcpyHostToDevice);
	checkCudaErrors(cudaStatus);	
	
	const dim3 block(BLOCKDIM,BLOCKDIM);
	const dim3 grid(input.cols/BLOCKDIM, input.rows/BLOCKDIM);

	median_filter_2d<<<grid,block>>>(d_input,d_output,input.cols,input.rows);

	cudaStatus = hipDeviceSynchronize();
	checkCudaErrors(cudaStatus);	

	cudaStatus = hipMemcpy(output.ptr(),d_output,output.rows*output.cols,hipMemcpyDeviceToHost);
	checkCudaErrors(cudaStatus);	

	cudaStatus = hipFree(d_input);
	checkCudaErrors(cudaStatus);	
	cudaStatus = hipFree(d_output);
	checkCudaErrors(cudaStatus);	
}


