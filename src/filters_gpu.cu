#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

__device__ void sort(float *x, int n_size) {
	// iterate over reference vector
	for (int i = 0; i < n_size-1; i++) {
		// initialize minimum element index
		int min_idx = i;
		// compare against rest of the elements
		for (int j = i + 1; j < n_size; j++) {
			// comparison
			if(x[j] < x[min_idx])
				min_idx = j;
		}
		// swap elements with minimum element
		float temp = x[min_idx];
		x[min_idx] = x[i];
		x[i] = temp;
	}	
}

__global__ void median_filter_2d(float *image_in, float *image_out,
				int size, int dim_2, int dim_3, 
				int kernel_size_r)
{
	// find thread id in global memory organization
	int thread_id = threadIdx.x + (blockDim.x * blockIdx.x);
	// if within image limits (max size)
	if (thread_id < size) {
		// find x and y indices
		int x = thread_id % dim_3; // dim3 is the size of the row
		int y = thread_id / dim_3; // equivalently #cols * size
		float xs[11*11]; // allocate some memory for presort
		int xs_size = 0;
		// iterate over image x axis
		for (int x_iter = x - kernel_size_r; x_iter <= x + kernel_size_r; x_iter ++) {
			// iterate over image y axis
			for (int y_iter = y - kernel_size_r; y_iter <= y + kernel_size_r; y_iter++) {
				// stay within image block dimensions
				if (0<=x_iter && x_iter < dim_3 && 0 <= y_iter && y_iter < dim_2) {
					// fill up pre-sorted vector
					// image_in[row_offset*row_size + col_offset]
					xs[xs_size++] = image_in[y_iter * dim_3 + x_iter];
				}
			}
		}
		// sort the given vector using the device method sort(*x,n)
		sort(xs,xs_size);
		// allocate the median of the sorted vector to the pixel at image out
		image_out[thread_id] = xs[xs_size/2];
	}
}


__global__ void cvrgb_to_gray(unsigned char* input, unsigned char* output, int width, int height, int colorWidthStep, int grayWidthStep)
{
	const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

	//Only valid threads perform memory I/O
	if((xIndex<width) && (yIndex<height))
	{
		const int color_tid = yIndex * colorWidthStep + xIndex;
		const int gray_tid  = yIndex * grayWidthStep + xIndex;
		const float gray = input[color_tid];
		output[gray_tid] = static_cast<unsigned char>(gray);
	}
}

void filter_wrapper(const cv::Mat& input, cv::Mat& output)
{
	const int colorBytes = input.step * input.rows;
	const int grayBytes = output.step * output.rows;
	unsigned char *d_input, *d_output;

	printf("ColorBytes = %d | grayBytes = %d\n",colorBytes,grayBytes);
	
	hipError_t cudaStatus;	
	
	cudaStatus = hipMalloc<unsigned char>(&d_input,colorBytes);
	checkCudaErrors(cudaStatus);	
	cudaStatus = hipMalloc<unsigned char>(&d_output,grayBytes);
	checkCudaErrors(cudaStatus);

	cudaStatus = hipMemcpy(d_input,input.ptr(),colorBytes,hipMemcpyHostToDevice);
	checkCudaErrors(cudaStatus);	
	
	const dim3 block(16,16);

	const dim3 grid((input.cols + block.x - 1)/block.x, (input.rows + block.y - 1)/block.y);

	const int colwidstep = input.step;
	const int graywidstep = output.step;

	printf("Color Width Step = %d | Gray Width Step = %d \n",colwidstep,graywidstep);
	cvrgb_to_gray<<<grid,block>>>(d_input,d_output,input.cols,input.rows,colwidstep,graywidstep);

	//median_filter_2d<<<grid,block>>>(d_input,d_output,input.cols,input.rows,input.step,output.step);

	cudaStatus = hipDeviceSynchronize();
	checkCudaErrors(cudaStatus);	

	cudaStatus = hipMemcpy(output.ptr(),d_output,grayBytes,hipMemcpyDeviceToHost);
	checkCudaErrors(cudaStatus);	

	cudaStatus = hipFree(d_input);
	checkCudaErrors(cudaStatus);	
	cudaStatus = hipFree(d_output);
	checkCudaErrors(cudaStatus);	
}


