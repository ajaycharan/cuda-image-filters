#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

__device__ void sort(float *x, int n_size) {
	// iterate over reference vector
	for (int i = 0; i < n_size-1; i++) {
		// initialize minimum element index
		int min_idx = i;
		// compare against rest of the elements
		for (int j = i + 1; j < n_size; j++) {
			// comparison
			if(x[j] < x[min_idx])
				min_idx = j;
		}
		// swap elements with minimum element
		float temp = x[min_idx];
		x[min_idx] = x[i];
		x[i] = temp;
	}	
}

__global__ void median_filter_2d(unsigned char* input, unsigned char* output, int width, int height, int colorWidthStep, int grayWidthStep, int kernel_size_r)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if((x<width) && (y<height))
	{
		const int color_tid = y * colorWidthStep + x;
		float xs[11*11];
		int xs_size = 0;

		for (int x_iter = x - kernel_size_r; x_iter <= x + kernel_size_r; x_iter ++) {
			for (int y_iter = y - kernel_size_r; y_iter <= y + kernel_size_r; y_iter++) {
				if (0<=x_iter && x_iter < colorWidthStep && 0 <= y_iter && y_iter < height) {
					xs[xs_size++] = input[y_iter * colorWidthStep + x_iter];
				}
			}
		}
		sort(xs,xs_size);
		output[color_tid] = static_cast<unsigned char>(xs[xs_size/2]);
	}
}

void filter_wrapper(const cv::Mat& input, cv::Mat& output)
{
	const int colorBytes = input.step * input.rows;
	const int grayBytes = output.step * output.rows;
	unsigned char *d_input, *d_output;
	const int kernel = 5;

	//printf("ColorBytes = %d | grayBytes = %d\n",colorBytes,grayBytes);
	
	hipError_t cudaStatus;	
	
	cudaStatus = hipMalloc<unsigned char>(&d_input,colorBytes);
	checkCudaErrors(cudaStatus);	
	cudaStatus = hipMalloc<unsigned char>(&d_output,grayBytes);
	checkCudaErrors(cudaStatus);

	cudaStatus = hipMemcpy(d_input,input.ptr(),colorBytes,hipMemcpyHostToDevice);
	checkCudaErrors(cudaStatus);	
	
	const dim3 block(16,16);

	const dim3 grid((input.cols + block.x - 1)/block.x, (input.rows + block.y - 1)/block.y);

	const int colwidstep = input.step;
	const int graywidstep = output.step;

	//printf("Color Width Step = %d | Gray Width Step = %d \n",colwidstep,graywidstep);

	median_filter_2d<<<grid,block>>>(d_input,d_output,input.cols,input.rows,colwidstep,graywidstep,kernel);

	cudaStatus = hipDeviceSynchronize();
	checkCudaErrors(cudaStatus);	

	cudaStatus = hipMemcpy(output.ptr(),d_output,grayBytes,hipMemcpyDeviceToHost);
	checkCudaErrors(cudaStatus);	

	cudaStatus = hipFree(d_input);
	checkCudaErrors(cudaStatus);	
	cudaStatus = hipFree(d_output);
	checkCudaErrors(cudaStatus);	
}


